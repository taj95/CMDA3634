#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<time.h>
#include<math.h>

#include "hip/hip_runtime.h"

//device funcion
__global__ void kernalAddVectors(double *a, double *b, double*c) {
  
  int threadid = threadIdx.x; // thread number
  int blockid = blockIdx.x; // block number
  int Nblock = blockDim.x; //number of threads in a block
  
  int id = threadid + blockid*Nblock;

  if (id<N) {
    c[id] = a[id] + b[id];
  }
}

int main(int argc, char **argv) {

  // get vector size from command line argument
  int N = atoi(argv[1]);
  
  double seed = clock();
  srand48(seed);

 double *h_a, *h_b, *h_c; //host vectors

  //alocate storage
  h_a = (double *) malloc(N*sizeof(double));
  h_b = (double *) malloc(N*sizeof(double));
  h_c = (double *) malloc(N*sizeof(double));

  //populate a and b
  for(int n=0; n<N; n++) {
    h_a[n] = drand48();
    h_b[n] = drand48();
  }

  double hostStart = clock();

  // c=a+b
  for(int n=0;n<N:n++) {
    h_c[n] = h_a[n] + h_b[n];
  }

  double hostEnd = clock();
  double hostTime = (hostEnd - hostStart)/(double) CLOCKS_PER_SEC;

  printf("The host took %g seconds to add a and b \n", hostTime);

  //Device arrays
  double *d_a, *d_b, *d_c;

  //allocate memory on the Device with hipMalloc
  hipMalloc(&d_a, N*sizeof(double));
  hipMalloc(&d_b, N*sizeof(double));
  hipMalloc(&d_c, N*sizeof(double));

  //copy data from the host to the device
  hipMemcpy(d_a, h_a, N*sizeof(double), cuaMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N*sizeof(double), cuaMemcpyHostToDevice);
  
  //at his point the data is allocated and populated on the device

  int Nthreads = atoi(argv[2]); //get the num of threads per block
  int Nblocks = (N+Nthreads-1)/Nthreads;

  double deviceStart = clock(); 

  kernalAddVectors <<< , >>>(N, d_a, d_b, d_c);

  double deviceEnd = clock(); 
  double deivceTime = (deviceEnd-deviceStart)/(double) CLOCKS_PER_SEC;

  printf("The device took %f seconds to add a and b \n", deviceTime);
  printf("The device was %d times faster \n", hostTime/deviceTime);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);


  free(h_a);
  free(h_b);
  free(h_c);

}
