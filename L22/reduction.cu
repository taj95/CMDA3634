#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"

__global__ reduction(int N, float *a, float* result) {
	
	int thread = threadIdx.x;
	int block = blockIdx.x;
	int blockSize = blockDim.x;

	//unique global thread ID
	int id = thread + block*blockSize;


	__shared__ float s_sum[32];
	s_sum[id] = a[id]; //add the threads id to start
	__syncthreads(); //make sure the write to shared is finshed VERY IMPORTANT


	//float sum = a[id]
	if (id<16) {
		s_sum += s_sum[id+16];
	}

	__syncthreads();

	



	//float sum = 0;
	//for (int n=0; n<N; n++) {
	//	sum += a[n];
	//}
	//*result = sum;

}

//perform a reduction on a vector of length n
int main(int argc, char **argv) {

	int N = 32;
	double seed = 0;
	srand48(seed);

	//allocate memory on host
	float *h_a = (float*)malloc(N*sizeof(float));

	//populate with random data
	for (int n=0; n<N; n++) {
		h_a[n] = drand48();
	}

	//perform the reduction on host
	float h_sum = 0.;
	for (int n=0; n<N; n++) {
		h_sum += h_a[n];
	}

	printf("The Host's sum was %f \n", h_sum);

	float *d_a;
	hipMalloc(&d_a, N*sizeof(float));
	hipMalloc(&d_sum, 1*sizeof(float));

	//populate the device array with the same data as the host
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);

	//block dimensions
	dim3 B(32,1,1);

	//grid dimensions
	dim3 G((N+32-1)/32, 1, 1);

	reduction <<<G,B>>> (N, d_a, d_sum);

	hipMemcpy(&h_sum, d_sum, 1*sizeof(float), hipMemcpyDeviceToHost);

	printf("The Devic's sum was %f \n", h_sum);

}
