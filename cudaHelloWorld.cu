#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

// This is my deviece function
// __global__ means this function is visible to the host
__global__ void kernelHelloWorld() {
	printf("Hello World!\n");
}

int main(int argc, char** argv) {

	int Nblocks = 10; // number of blocks
	int Nthreads = 3; //number of threads per block

	// run the function "kernelHelloworld" on the device
	kernelHelloWorld <<< Nblocks, Nblocks >>> ();
}
